#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

int thread_num=800;

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *cu_img, int width, int maxIterations, int thread_num) {
    int x_idx, y_idx, tot;
    tot = blockIdx.x*thread_num+threadIdx.x;
    x_idx = tot%(width);
    y_idx = tot/(width);

    cu_img[tot] = mandel(lowerX + x_idx * stepX, lowerY + y_idx * stepY, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int tot_len_size = resX*resY*sizeof(int);

    int *value = (int*)malloc(tot_len_size);

    int *cu_img;
    hipMalloc((void**)&cu_img, tot_len_size);
    
    mandelKernel<<<resX*resY/thread_num, thread_num>>>(lowerX, lowerY, stepX, stepY, cu_img, resX, maxIterations, thread_num);

    hipMemcpy(value, cu_img, tot_len_size, hipMemcpyDeviceToHost);
    memcpy(img, value, tot_len_size);
}