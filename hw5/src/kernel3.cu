#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

int thread_num=200;
int work_size=2;

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *cu_img, int width, int maxIterations, int thread_num, size_t pitch_img, int work_size) {
    int x_idx, y_idx, tot;
    tot = (blockIdx.x*thread_num+threadIdx.x)*work_size;

    float x, y;
    x_idx = tot%(width);
    y_idx = tot/(width);
    x = lowerX + x_idx * stepX;
    y = lowerY + y_idx * stepY;
    for(int idx=work_size-1; idx>0; idx--){
        *((int*)((char*)cu_img+(y_idx*pitch_img))+x_idx) = mandel(x, y, maxIterations);
        x_idx += 1;
        if(x_idx == width){
            x_idx = 0;
            y_idx += 1;
            x = lowerX;
            y += stepY;
        }
        else
            x = lowerX + x_idx * stepX;
    }
    *((int*)((char*)cu_img+(y_idx*pitch_img))+x_idx) = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int tot_len = resX*resY;
    int tot_len_size = resX*resY*sizeof(int);
    int resXsize = resX*sizeof(int);

    int *value;
    hipHostAlloc((void**)&value, tot_len_size, hipHostMallocDefault);

    int *cu_img;
    size_t pitch_img;
    hipMallocPitch((void**)&cu_img, &pitch_img, resXsize, resY);
    
    mandelKernel<<<tot_len/thread_num/work_size, thread_num>>>(lowerX, lowerY, stepX, stepY, cu_img, resX, maxIterations, thread_num, pitch_img, work_size);

    hipMemcpy2D(value, resXsize, cu_img, pitch_img, resXsize, resY, hipMemcpyDeviceToHost);
    memcpy(img, value, tot_len_size);
}